#include "hip/hip_runtime.h"
#include "methods_cuda.h"

__global__ void solve_level_csr(int start, int end, int *levelSet, int *Lp, int *Li, double *Lx, double *x)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx + start < end)
    {
        
        int i = levelSet[idx + start];
        for (int j = Lp[i]; j < Lp[i + 1] - 1; j++)
        {
            x[i] -= Lx[j] * x[Li[j]];
        }
        x[i] /= Lx[Lp[i + 1] - 1];
        
    }

}

extern "C" void sptrsv_csr_levelset_cuda(int n, int Lnz, int* Lp, int* Li, double* Lx,
    double* x,
    int levels, int* levelPtr,
    int* levelSet)
{
   
    
    int *d_Lp, *d_Li, *d_levelSet;
    double *d_Lx, *d_x;
    hipMalloc((void **)&d_Lp, (n + 1) * sizeof(int));
    hipMalloc((void **)&d_Li, Lnz * sizeof(int));
    hipMalloc((void **)&d_levelSet, n * sizeof(int));
    hipMalloc((void **)&d_Lx, Lnz * sizeof(double));
    hipMalloc((void **)&d_x, n * sizeof(double));
    hipMemcpy(d_Lp, Lp, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Li, Li, Lnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_levelSet, levelSet, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Lx, Lx, Lnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(double), hipMemcpyHostToDevice);
    
    for (int l = 0; l < levels; l++)
    {
        int start = levelPtr[l];
        int end = levelPtr[l + 1];
        int Blocks = (end - start - 1) / THREAD_PER_BLOCK + 1;
        solve_level_csr<<<Blocks,THREAD_PER_BLOCK>>>(start, end, d_levelSet, d_Lp, d_Li, d_Lx, d_x);
        
    }
    
    hipMemcpy(x, d_x, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_Lp);
    hipFree(d_Li);
    hipFree(d_levelSet);
    hipFree(d_Lx);
    hipFree(d_x);
    

}

__global__ void  add_row(int matrix_size, double* A_row, double* B_row, double* C_row)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < matrix_size)
    {
        C_row[idx] = A_row[idx] + B_row[idx];
    }
    
}

void matrix_add_cuda(int matrix_size, double **A, double **B, double **C)
{
    double *A_row, *B_row, *C_row;
    hipMalloc((void **)&A_row, matrix_size * sizeof(double));
    hipMalloc((void **)&B_row, matrix_size * sizeof(double));
    hipMalloc((void **)&C_row, matrix_size * sizeof(double));
    for (int i = 0; i < matrix_size; i++)
    {
        hipMemcpy(A_row, A[i], matrix_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_row, B[i], matrix_size * sizeof(double), hipMemcpyHostToDevice);

        int Blocks = (matrix_size - 1) / THREAD_PER_BLOCK + 1;
        add_row<<<Blocks,THREAD_PER_BLOCK>>>(matrix_size, A_row, B_row, C_row);

        hipMemcpy(C[i], C_row, matrix_size * sizeof(double), hipMemcpyDeviceToHost);
    }
   hipFree(A_row);
   hipFree(B_row);
   hipFree(C_row);

}

void vector_add_cuda(int vector_size, double *A, double *B, double *C)
{
    double *A_row, *B_row, *C_row;
    hipMalloc((void **)&A_row, vector_size * sizeof(double));
    hipMalloc((void **)&B_row, vector_size * sizeof(double));
    hipMalloc((void **)&C_row, vector_size * sizeof(double));

    hipMemcpy(A_row, A, vector_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_row, B, vector_size * sizeof(double), hipMemcpyHostToDevice);

    int Blocks = (vector_size - 1) / THREAD_PER_BLOCK + 1;
    add_row<<<Blocks, THREAD_PER_BLOCK>>>(vector_size, A_row, B_row, C_row);

    hipMemcpy(C, C_row, vector_size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(A_row);
    hipFree(B_row);
    hipFree(C_row);
}

__global__ void
spmv_csr_vector_kernel(const int num_rows,
                       const int *ptr,
                       const int *indices,
                       const double *data,
                       const double *x,
                       double *y)
{
    __shared__ volatile double vals[1024];
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x; // global thread index
    int warp_id = thread_id / 32;                          // global warp index
    int lane = thread_id & (32 - 1);                       // thread index within the warp
    // each warp process one row
    int row = warp_id;
    vals[threadIdx.x] = 0;
    
    if (row < num_rows)
    {
        int row_start = ptr[row];
        int row_end = ptr[row + 1];
        // compute running sum per thread
        for (int jj = row_start + lane; jj < row_end; jj += 32)
            vals[threadIdx.x] += data[jj] * x[indices[jj]];

        // reduction within warp to the first thread
        if (lane < 16)
            vals[threadIdx.x] += vals[threadIdx.x + 16];
        if (lane < 8)
            vals[threadIdx.x] += vals[threadIdx.x + 8];
        if (lane < 4)
            vals[threadIdx.x] += vals[threadIdx.x + 4];
        if (lane < 2)
            vals[threadIdx.x] += vals[threadIdx.x + 2];
        if (lane < 1)
            vals[threadIdx.x] += vals[threadIdx.x + 1];
        // first thread writes the result
        if (lane == 0)
            y[row] += vals[threadIdx.x];
            
    }
}

void spmv_csr_cuda(int n, int *Ap, int *Ai, double *Ax, double *x, double *y)
{
    int nz = Ap[n];
    int *csrRowPtrA, *csrColIndA;
    double *valA, *valx, *valy; 
    hipMalloc((void **)&csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void **)&csrColIndA, nz * sizeof(int));
    hipMalloc((void **)&valA, nz * sizeof(double));
    hipMalloc((void **)&valx, n * sizeof(double));
    hipMalloc((void **)&valy, n * sizeof(double));
    hipMemcpy(csrRowPtrA, Ap, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(csrColIndA, Ai, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(valA, Ax, nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valx, x, n * sizeof(double), hipMemcpyHostToDevice);

    int num_threads = n * 32; // Each warp (32 threads) takes one row
    int Blocks = (num_threads - 1) / THREAD_PER_BLOCK + 1;
    spmv_csr_vector_kernel<<<Blocks, THREAD_PER_BLOCK>>>(n, csrRowPtrA, csrColIndA, valA, valx, valy);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    hipMemcpy(y, valy, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(csrRowPtrA);
    hipFree(csrColIndA);
    hipFree(valA);
    hipFree(valx);
    hipFree(valy);
}