#include "methods_cusparse.h"
enum DATA_DIRECTION{
    row_major = 0,
    column_major = 1,
};

void spmv_csr_cusparse(int n, int *Ap, int *Ai, double *Ax, double *x, double *y)
{
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseStatus_t status;

    float transfer_in, computation_time, transfer_out; // timing values
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int nz = Ap[n];
    int *csrRowPtrA, *csrColIndA;
    double *valA, *valx, *valy; 
    hipEventRecord(start);
    hipMalloc((void **)&csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void **)&csrColIndA, nz * sizeof(int));
    hipMalloc((void **)&valA, nz * sizeof(double));
    hipMalloc((void **)&valx, n * sizeof(double));
    hipMalloc((void **)&valy, n * sizeof(double));
    hipMemcpy(csrRowPtrA, Ap, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(csrColIndA, Ai, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(valA, Ax, nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valx, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valy, y, n * sizeof(double), hipMemcpyHostToDevice);
	
	hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_in, start, stop);

    hipEventRecord(start);

    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;

    hipsparseCreateCsr(&matA, n, n, nz,
                      csrRowPtrA, csrColIndA, valA,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    
    hipsparseCreateDnVec(&vecX, n, valx, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, n, valy, HIP_R_64F);

    double alpha = 1.0;
    double beta  = 0.0;
    size_t buffersize = 0;
    status = hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_CSRMV_ALG1, &buffersize);
    
    void *buffer;
    hipMalloc(&buffer, buffersize);

    status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_CSRMV_ALG1, buffer);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);

    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("CuSparse csrmv failed.\n");
        exit(-1);
    }
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);

    hipEventRecord(start);
    hipMemcpy(y, valy, n * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_out, start, stop);
    // print timing results
    printf("%15.5f %15.5f %15.5f\n", transfer_in,
            computation_time, transfer_out);

    hipFree(buffer);
    hipFree(csrRowPtrA);
    hipFree(csrColIndA);
    hipFree(valA);
    hipFree(valx);
    hipFree(valy);
}

void spmv_bcsr_cusparse(int n, int nb, int blockDim, int *Bp, int *Bi, double *Bx, double *x, double *y, int direction)
{
    int num_block = Bp[nb];
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;
    if (direction == column_major)
    {
        dir = HIPSPARSE_DIRECTION_COLUMN;
    }
    

    float transfer_in, computation_time, transfer_out;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *bcsrRowPtr, *bcsrColInd;
    double *bcsrVal, *valx, *valy; 
    hipEventRecord(start);
    hipMalloc((void **)&bcsrRowPtr, (nb + 1) * sizeof(int));
    hipMalloc((void **)&bcsrColInd, num_block * sizeof(int));
    hipMalloc((void **)&bcsrVal, num_block * blockDim * blockDim * sizeof(double));
    hipMalloc((void **)&valx, nb * blockDim * sizeof(double));
    hipMalloc((void **)&valy, nb * blockDim * sizeof(double));
    hipMemcpy(bcsrRowPtr, Bp, (nb + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bcsrColInd, Bi, num_block * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bcsrVal, Bx, num_block * blockDim * blockDim * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valx, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valy, y, n * sizeof(double), hipMemcpyHostToDevice);

	hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_in, start, stop);

    hipEventRecord(start);
    double alpha = 1.0;
    double beta  = 0.0;
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    
    status = hipsparseDbsrmv(handle, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        nb, nb, num_block, &alpha, 
        descr, bcsrVal, bcsrRowPtr, bcsrColInd, blockDim, valx, &beta, valy);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);

    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("CuSparse bsrmv failed.\n");
        exit(-1);
    }

    hipEventRecord(start);
    hipMemcpy(y, valy, n * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&transfer_out, start, stop);
    // print timing results
    printf("%15.5f %15.5f %15.5f\n", transfer_in,
            computation_time, transfer_out);

    hipFree(bcsrRowPtr);
    hipFree(bcsrColInd);
    hipFree(bcsrVal);
    hipFree(valx);
    hipFree(valy);
}