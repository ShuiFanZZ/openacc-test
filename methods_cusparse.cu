#include "methods_cusparse.h"

void spmv_csr_cusparse(int n, int *Ap, int *Ai, double *Ax, double *x, double *y)
{
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseStatus_t status;

    int nz = Ap[n];
    int *csrRowPtrA, *csrColIndA;
    double *valA, *valx, *valy; 
    hipMalloc((void **)&csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void **)&csrColIndA, nz * sizeof(int));
    hipMalloc((void **)&valA, nz * sizeof(double));
    hipMalloc((void **)&valx, n * sizeof(double));
    hipMalloc((void **)&valy, n * sizeof(double));
    hipMemcpy(csrRowPtrA, Ap, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(csrColIndA, Ai, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(valA, Ax, nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valx, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(valy, y, n * sizeof(double), hipMemcpyHostToDevice);

    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;

    hipsparseCreateCsr(&matA, n, n, nz,
                      csrRowPtrA, csrColIndA, valA,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    
    hipsparseCreateDnVec(&vecX, n, valx, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, n, valy, HIP_R_64F);

    double alpha = 1.0;
    double beta  = 0.0;
    size_t buffersize = 0;
    status = hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_CSRMV_ALG1, &buffersize);
    
    void *buffer;
    hipMalloc(&buffer, buffersize);

    status = hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_CSRMV_ALG1, buffer);

    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("CuSparse csrmv failed.\n");
        exit(-1);
    }
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);

    
    hipMemcpy(y, valy, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(buffer);
    hipFree(csrRowPtrA);
    hipFree(csrColIndA);
    hipFree(valA);
    hipFree(valx);
    hipFree(valy);
}